#include "hip/hip_runtime.h"
#include "bf16_fused_adam.h"

#include <ATen/core/Tensor.h>
#include <ATen/native/cuda/ForeachFunctors.cuh>
#include <ATen/native/cuda/MultiTensorApply.cuh>
#include <ATen/native/cuda/Pow.cuh>
#include <utility>


namespace bf16_fused_adam {

constexpr int kArgsDepth = 5;

constexpr uint8_t kParamIdx = 0;
constexpr uint8_t kGradIdx = 1;
constexpr uint8_t kExpAvgIdx = 2;
constexpr uint8_t kExpAvgSqIdx = 3;
constexpr uint8_t kMantissaIdx = 4;

template <typename T>
__device__ __forceinline__ T lerp(const T v0, const T v1, const T t) {
    return fma(t, v1, fma(-t, v0, v0));
}

__device__ __forceinline__ float concat_float(const at::BFloat16 value, const at::BFloat16 mantissa) {
    return reinterpret_cast<float>(
        (static_cast<uint32_t>(reinterpret_cast<uint16_t>(value)) << 16) | 
         static_cast<uint32_t>(reinterpret_cast<uint16_t>(mantissa))
    );
}

__device__ __forceinline__ void split_float(const float f, at::BFloat16 &value, at::BFloat16 &mantissa) {
    value = reinterpret_cast<at::BFloat16>(static_cast<uint16_t>(reinterpret_cast<uint32_t>(f) >> 16));
    mantissa = reinterpret_cast<at::BFloat16>(static_cast<uint16_t>(reinterpret_cast<uint32_t>(f)));
}

__device__ __forceinline__ void adamw_math(
    at::BFloat16 r_args[kArgsDepth][kILP],
    const float &step_size,
    const float &wd_step_size,
    const float &beta1,
    const float &beta2,
    const float &weight_decay,
    const float &eps,
    const float &bias_correction2_sqrt)
{
#pragma unroll
    for (int ii = 0; ii < kILP; ii++)
    {
        // Load values.
        const float grad = static_cast<float>(r_args[kGradIdx][ii]);

        float param = concat_float(r_args[kParamIdx][ii], r_args[kMantissaIdx][ii]);

        float exp_avg = static_cast<float>(r_args[kExpAvgIdx][ii]);
        float exp_avg_sq = static_cast<float>(r_args[kExpAvgSqIdx][ii]);

        param -= wd_step_size * param;

        exp_avg = lerp(grad, exp_avg, beta1);
        exp_avg_sq = lerp(grad * grad, exp_avg_sq, beta2);

        const float denom = (std::sqrt(exp_avg_sq) / bias_correction2_sqrt) + eps;
        param -= step_size * exp_avg / denom;

        // Store results.
        split_float(param, r_args[kParamIdx][ii], r_args[kMantissaIdx][ii]);
        r_args[kExpAvgIdx][ii] = exp_avg;
        r_args[kExpAvgSqIdx][ii] = exp_avg_sq;
    }
}

struct FusedAdamMathFunctor {
  __device__ __forceinline__ void operator()(
      int chunk_size,
      FusedOptimizerTensorListMetadata<kArgsDepth>& tl,
      const double& lr,
      const double& beta1,
      const double& beta2,
      const double& weight_decay,
      const double& eps) {
    const auto tensor_loc = tl.block_to_tensor[blockIdx.x];
    const auto chunk_idx = tl.block_to_chunk[blockIdx.x];

    const auto [step_size, wd_step_size, bias_correction2_sqrt] =
        [&]() -> std::tuple<double, double> {
      auto* step_count = reinterpret_cast<const float*>(tl.state_steps_addresses[tensor_loc]);
      const auto bias_correction1 = 1 - at::native::pow_(beta1, *step_count);
      const auto bias_correction2 = 1 - at::native::pow_(beta2, *step_count);
      const auto bias_correction2_sqrt = std::sqrt(bias_correction2);
      return {lr * bias_correction1, lr * weight_decay, bias_correction2_sqrt};
    }();

    at::BFloat16* args[kArgsDepth];
    at::BFloat16 r_args[kArgsDepth][kILP];
    const auto n = tl.numel_for_tensor[tensor_loc] - chunk_idx * chunk_size;

    const bool all_aligned{
        init_args<kArgsDepth>(args, tl, chunk_idx, chunk_size, tensor_loc)};
    if ((n % kILP == 0) && (chunk_size % kILP == 0) && all_aligned) {
      for (int64_t i_start = threadIdx.x;
           i_start * kILP < n && i_start * kILP < chunk_size;
           i_start += blockDim.x) {
#pragma unroll
        for (int i = 0; i < kArgsDepth; i++) {
          load_store(r_args[i], args[i], 0, i_start);
        }
        adam_math(
            r_args,
            step_size,
            wd_step_size,
            beta1,
            beta2,
            weight_decay,
            eps,
            bias_correction2_sqrt);
#pragma unroll
        for (int i = 0; i < kArgsDepth; i++) {
          if (i != kGradIdx) {
            load_store(args[i], r_args[i], i_start, 0);
          }
        }
      }
    } else {
      for (int64_t i_start = 0; i_start < n && i_start < chunk_size;
           i_start += blockDim.x * kILP) {
        load_args<kArgsDepth>(r_args, args, i_start, chunk_size, n);
        adam_math(
            r_args,
            step_size,
            wd_step_size,
            beta1,
            beta2,
            weight_decay,
            eps,
            bias_correction2_sqrt);
#pragma unroll
        for (int i = 0; i < kArgsDepth; i++) {
          if (i != kGradIdx) {
            store_args(args[i], r_args[i], i_start, chunk_size, n);
          }
        }
      }
    }
  }
};

void bf16_fused_adamw_cuda_impl_(
    at::TensorList params,
    at::TensorList grads,
    at::TensorList exp_avgs,
    at::TensorList exp_avg_sqs,
    at::TensorList mantissas,
    at::TensorList state_steps,
    const double lr,
    const double beta1,
    const double beta2,
    const double weight_decay,
    const double eps) {
  std::vector<std::vector<at::Tensor>> tensor_lists{
      params.vec(), grads.vec(), exp_avgs.vec(), exp_avg_sqs.vec(), mantissas.vec()};

  AT_DISPATCH_FLOATING_TYPES_AND(
      kBFloat16,
      params[0].scalar_type(),
      "bf16_fused_adamw_kernel_cuda",
      [&]() {
        multi_tensor_apply_for_fused_optimizer<5>(
            tensor_lists,
            state_steps,
            FusedAdamMathFunctor(),
            lr,
            beta1,
            beta2,
            weight_decay,
            eps);
      });
}

} // namespace bf16_fused_adam